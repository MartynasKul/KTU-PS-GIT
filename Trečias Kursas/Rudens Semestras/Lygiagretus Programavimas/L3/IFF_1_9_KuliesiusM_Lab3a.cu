#include "hip/hip_runtime.h"
%%cu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
//kuo skiriasi device nuo global? = device funkcijos naudojasi tik global funkcijoje, kitur ju naudot negalima, jos vykdomos ant GPU, gali kviesti tik kitos funkcijos kurios naudoja GPU
// Device -funkcija kuri gali buti kvieciama ir vykdoma per GPU, negali buti kvieciama is CPU
// global funkcijos gali buti kvieciamos per CPU ir vykdomos tik per GPU

// Define the structure for an entry
struct Entry {
    char word[256];
    int randomiseamount;
    float filterdouble;
};


__device__ bool isAlphabet(char c) {
    return (c >= 'a' && c <= 'z') || (c >= 'A' && c <= 'Z');
}

__device__ bool isUpperCase(char c) {
    return (c >= 'A' && c <= 'Z');
}
// Vykdomas duomenu apdorojimas su globalia funkcija
__global__ void processDataKernel(Entry* entries, Entry* results, int dataSize) {
    int numThreads = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = numThreads;

    for (int threadIndex = tid; threadIndex < dataSize; threadIndex += stride) {
        char* word = entries[threadIndex].word;
        int wordLength = 0;

        while (word[wordLength] != '\0') {
            ++wordLength;
        }

        // patikrina ar susiapvalina i didesne puse, jei taip - irasomas irasas.
        if (static_cast<int>(entries[threadIndex].filterdouble + 0.5) > entries[threadIndex].filterdouble){
            for (int i = 0; i < wordLength; ++i) {
                if (isAlphabet(word[i])) {
                    char base = isUpperCase(word[i]) ? 'A' : 'a';
                    results[threadIndex].word[i] = static_cast<char>((static_cast<int>(word[i]) + entries[threadIndex].randomiseamount - static_cast<int>(base)) % 26 + static_cast<int>(base));
                }
            }
        }
    }
}


int main() {
    // Read data from the CSV file
    //std::ifstream inputFile("IFF19KuliesiusMartynas1.csv"); //puse atitinka
    //std::ifstream inputFile("IFF19KuliesiusMartynas2.csv"); //visi neatitinka
    std::ifstream inputFile("IFF19KuliesiusMartynas3.csv"); //visi atitinka
    if (!inputFile.is_open()) {
        std::cerr << "Error opening the input file." << std::endl;
        return 1;
    }

    // Convert CSV data to a vector of entries
    // Skip the header row
    std::string header;
    std::getline(inputFile, header);

    // Convert CSV data to a vector of entries
    std::vector<Entry> entries;
    std::string line;;
    while (std::getline(inputFile, line)) {
        std::istringstream ss(line);
        std::string token;

        Entry entry;

        // Read word
        std::getline(ss, token, ',');
        strncpy(entry.word, token.c_str(), sizeof(entry.word) - 1);
        entry.word[sizeof(entry.word) - 1] = '\0';  // Ensure null-termination

        // Read randomiseamount
        std::getline(ss, token, ',');
        entry.randomiseamount = std::stoi(token);

        // Read filterdouble
        std::getline(ss, token, ',');
        entry.filterdouble = std::stof(token);

        entries.push_back(entry);
    }

    inputFile.close();

    // Determine the size of data
    int dataSize = entries.size();
    if (dataSize == 0) {
        std::cerr << "No data found in the input file." << std::endl;
        return 1;
    }

    // Allocate memory for results on the host
    Entry* results = new Entry[dataSize];


    //Cuda configuration
    int threadsPerBlock = 2;
    int blocksPerGrid = (dataSize + threadsPerBlock) / threadsPerBlock;
    std::cout << "Total number of blocks on the grid: " << blocksPerGrid << std::endl;
    int totalThreads = blocksPerGrid * threadsPerBlock;
    std::cout << "Total number of threads: " << totalThreads << std::endl;
    dim3 blocks(blocksPerGrid, 1, 1);
    dim3 threads(threadsPerBlock, 1, 1);


    // Allocate memory for entries and results on the device
    Entry* d_entries;
    Entry* d_results;
    hipMalloc((void**)&d_entries, dataSize * sizeof(Entry)); // kodel prie void dvi ** = kad duomenys pointerio grazintusi tarp metodu
    hipMalloc((void**)&d_results, dataSize * sizeof(Entry));


    // Copy input data from CPU to GPU
    hipMemcpy(d_entries, entries.data(), dataSize * sizeof(Entry), hipMemcpyHostToDevice);
    hipMemcpy(d_results, results, dataSize * sizeof(Entry), hipMemcpyHostToDevice);


    // Launch the CUDA kernel
    processDataKernel <<<2, 32 >>> (d_entries, d_results, dataSize);

    // Copy results from GPU to CPU
    hipMemcpy(results, d_results, dataSize * sizeof(Entry), hipMemcpyDeviceToHost);

    std::ofstream outputFile("output.txt");
    if (!outputFile.is_open()) {
        std::cerr << "Error opening the output file." << std::endl;
        return 1;
    }
    for (int i = 0; i < dataSize; ++i) {
        if (results[i].word[0] != ' ' && results[i].word[0] != '\0')
        {
            outputFile << results[i].word << std::endl;
        }
    }
    // Write results to the file


    // Close the file
    outputFile.close();

    // Atlaisvinam atminti
    delete[] results;
    hipFree(d_entries);
    hipFree(d_results); // jei nenaudojam cuda free, alocated atmintis isliks GPU iki kol programa pabaigs darba, taip padidina atminties sunaudojima ir gali paveikti programos veikima (performance)

    return 0;
}



